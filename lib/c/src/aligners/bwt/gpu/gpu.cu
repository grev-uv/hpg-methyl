#include "hip/hip_runtime.h"
#include "gpu.cuh"

__device__ SA_TYPE getOcompValueGPU(SA_TYPE n, SA_TYPE m, comp_matrix O) {
	SA_TYPE pos, desp;
	pos  = m / FM_COMP_VALUE;
	desp = m % FM_COMP_VALUE;
#if   defined FM_COMP_32
	return O.desp[n][pos] + __popc  ( O.count[n][pos] << (FM_COMP_VALUE - (desp + 1)) );
#elif defined FM_COMP_64
	return O.desp[n][pos] + __popcll( O.count[n][pos] << (FM_COMP_VALUE - (desp + 1)) );
#endif
}

void reverse_strand_gpu_O(comp_matrix *r_O, comp_matrix *s_O) {

	hipError_t error;

	r_O->siz = s_O->siz;

	r_O->n_desp = s_O->n_desp;
	r_O->m_desp = s_O->m_desp;

	SA_TYPE **r_desp = (SA_TYPE **) malloc(r_O->n_desp * sizeof(SA_TYPE *));
	check_malloc(r_desp, "reverse_strand_O");
	SA_TYPE **s_desp = (SA_TYPE **) malloc(s_O->n_desp * sizeof(SA_TYPE *));
	check_malloc(s_desp, "reverse_strand_O");

	hipMemcpy(s_desp, s_O->desp, s_O->n_desp * sizeof(SA_TYPE *), hipMemcpyDeviceToHost);
	manageCudaError();

	if (AA != ((uint8_t) -1) && TT !=((uint8_t) -1)) {
		r_desp[AA] = s_desp[TT];
		r_desp[TT] = s_desp[AA];
	} else if (AA != ((uint8_t) -1)) {
		r_desp[AA] = s_desp[AA];
	} else if (TT != ((uint8_t) -1)) {
		r_desp[TT] = s_desp[TT];
	}

	if (CC != ((uint8_t) -1) && GG !=((uint8_t) -1)) {
		r_desp[CC] = s_desp[GG];
		r_desp[GG] = s_desp[CC];
	} else if (CC != ((uint8_t) -1)) {
		r_desp[CC] = s_desp[CC];
	} else if (GG != ((uint8_t) -1)) {
		r_desp[GG] = s_desp[GG];
	}

	hipMalloc((void**) &r_O->desp, r_O->n_desp * sizeof(SA_TYPE *));
	manageCudaError();
	hipMemcpy(r_O->desp, r_desp, r_O->n_desp * sizeof(SA_TYPE *), hipMemcpyHostToDevice);
	manageCudaError();

	free(r_desp);
	free(s_desp);

#if defined FM_COMP_32 || FM_COMP_64

	r_O->n_count = s_O->n_count;
	r_O->m_count = s_O->m_count;

	FM_COMP_TYPE **r_count = (FM_COMP_TYPE **) malloc(r_O->n_count * sizeof(FM_COMP_TYPE *));
	check_malloc(r_count, "reverse_strand_O");
	FM_COMP_TYPE **s_count = (FM_COMP_TYPE **) malloc(s_O->n_count * sizeof(FM_COMP_TYPE *));
	check_malloc(s_count, "reverse_strand_O");

	hipMemcpy(s_count, s_O->count, s_O->n_count * sizeof(FM_COMP_TYPE *), hipMemcpyDeviceToHost);
	manageCudaError();

	if (AA != ((uint8_t) -1) && TT !=((uint8_t) -1)) {
		r_count[AA] = s_count[TT];
		r_count[TT] = s_count[AA];
	} else if (AA != ((uint8_t) -1)) {
		r_count[AA] = s_count[AA];
	} else if (TT != ((uint8_t) -1)) {
		r_count[TT] = s_count[TT];
	}

	if (CC != ((uint8_t) -1) && GG !=((uint8_t) -1)) {
		r_count[CC] = s_count[GG];
		r_count[GG] = s_count[CC];
	} else if (CC != ((uint8_t) -1)) {
		r_count[CC] = s_count[CC];
	} else if (GG != ((uint8_t) -1)) {
		r_count[GG] = s_count[GG];
	}

	hipMalloc((void**) &r_O->count, r_O->n_count * sizeof(FM_COMP_TYPE *));
	manageCudaError();
	hipMemcpy(r_O->count, r_count, r_O->n_count * sizeof(FM_COMP_TYPE *), hipMemcpyHostToDevice);
	manageCudaError();

	free(r_count);
	free(s_count);

#endif

}

void read_comp_matrix_gpu(comp_matrix *matrix, const char *directory, const char *name) {

	size_t err=0;
	hipError_t error;
	FILE *fp;

	char path[500];

	path[0]='\0';
	strcat(path, directory);
	strcat(path, "/");
	strcat(path, name);
	strcat(path, ".desp");

	fp  = fopen(path,  "rb+");
	check_file_open(fp, path);

	err = fread(&matrix->siz,    sizeof(SA_TYPE),  1, fp);
	check_file_read(err, 1, path);

	err = fread(&matrix->n_desp, sizeof(SA_TYPE),  1, fp);
	check_file_read(err, 1, path);

	err = fread(&matrix->m_desp, sizeof(SA_TYPE),  1, fp);
	check_file_read(err, 1, path);

	hipHostMalloc((void**) &matrix->desp, matrix->n_desp * sizeof(SA_TYPE *));
	manageCudaError();

	for (SA_TYPE i=0; i<matrix->n_desp; i++) {
		hipHostMalloc((void**) &matrix->desp[i], matrix->m_desp * sizeof(SA_TYPE));
		manageCudaError();
		err = fread(matrix->desp[i], sizeof(SA_TYPE), matrix->m_desp, fp);
		check_file_read(err, matrix->m_desp, path);
	}

	fclose(fp);

#if defined FM_COMP_32 || FM_COMP_64
	path[0]='\0';
	strcat(path, directory);
	strcat(path, "/");
	strcat(path, name);
	strcat(path, ".count");

	fp  = fopen(path,  "rb+");
	check_file_open(fp, path);

	err = fread(&matrix->n_count,   sizeof(SA_TYPE),  1, fp);
	check_file_read(err, 1, path);

	err = fread(&matrix->m_count,   sizeof(SA_TYPE),  1, fp);
	check_file_read(err, 1, path);

	hipHostMalloc((void**) &matrix->count, matrix->n_count * sizeof(FM_COMP_TYPE *));
	manageCudaError();

	for (SA_TYPE i=0; i<matrix->n_count; i++){
		hipHostMalloc((void**) &matrix->count[i], matrix->m_count * sizeof(FM_COMP_TYPE));
		manageCudaError();
		err = fread(matrix->count[i], sizeof(FM_COMP_TYPE), matrix->m_count, fp);
		check_file_read(err, matrix->m_count, path);
	}

	fclose(fp);
#endif

}

void copy_vector_gpu(vector *device, vector *host) {

	hipError_t error;

	device->n = host->n;

	hipMalloc((void**) &device->vector,  device->n * sizeof(SA_TYPE));
	manageCudaError();
	hipMemcpy(device->vector, host->vector, device->n * sizeof(SA_TYPE), hipMemcpyHostToDevice);
	manageCudaError();

}

void copy_comp_matrix_gpu(comp_matrix *device, comp_matrix *host) {

	hipError_t error;

	device->siz    = host->siz;
	device->n_desp = host->n_desp;
	device->m_desp = host->m_desp;

	SA_TYPE **desp = (SA_TYPE **) malloc(host->n_desp * sizeof(SA_TYPE *)); 
	check_malloc(desp, "reverse_strand_O");

	for (SA_TYPE i=0; i<device->n_desp; i++) {
		hipMalloc((void**) &desp[i], host->m_desp * sizeof(SA_TYPE));
		manageCudaError();
		hipMemcpy(desp[i], host->desp[i], host->m_desp * sizeof(SA_TYPE), hipMemcpyHostToDevice);
		manageCudaError();
	}

	hipMalloc((void**) &device->desp, host->n_desp * sizeof(SA_TYPE *));
	manageCudaError();
	hipMemcpy(device->desp, desp, host->n_desp * sizeof(SA_TYPE *), hipMemcpyHostToDevice);
	manageCudaError();
	free(desp);

#if defined FM_COMP_32 || FM_COMP_64

	device->n_count = host->n_count;
	device->m_count = host->m_count;

	FM_COMP_TYPE **count = (FM_COMP_TYPE **) malloc(host->n_count * sizeof(FM_COMP_TYPE *)); 
	check_malloc(count, "reverse_strand_O");

	for (SA_TYPE i=0; i<device->n_count; i++) {
		hipMalloc((void**) &count[i], host->m_count * sizeof(FM_COMP_TYPE));
		manageCudaError();
		hipMemcpy(count[i], host->count[i], host->m_count * sizeof(FM_COMP_TYPE), hipMemcpyHostToDevice);
		manageCudaError();
	}

	hipMalloc((void**) &device->count, host->n_count * sizeof(FM_COMP_TYPE *));
	manageCudaError();
	hipMemcpy(device->count, count, host->n_count * sizeof(FM_COMP_TYPE *), hipMemcpyHostToDevice);
	manageCudaError();
	free(count);

#endif

}

void free_comp_matrix_gpu_host(comp_matrix *reverse, comp_matrix *strand) {

	for (SA_TYPE i=0; i<strand->n_desp; i++) {
		hipHostFree(strand->desp[i]);
#if defined FM_COMP_32 || FM_COMP_64
		hipHostFree(strand->count[i]);
#endif
	}

	hipHostFree(strand->desp);
	if (reverse != NULL) hipHostFree(reverse->desp);
#if defined FM_COMP_32 || FM_COMP_64
	hipHostFree(strand->count);
	if (reverse != NULL) hipHostFree(reverse->count);
#endif

}

void free_comp_matrix_gpu_device(comp_matrix *reverse, comp_matrix *strand) {

	hipError_t error;

	SA_TYPE **desp = (SA_TYPE **) malloc(strand->n_desp * sizeof(SA_TYPE *)); 
	check_malloc(desp, "reverse_strand_O");
	hipMemcpy(desp, strand->desp, strand->n_desp * sizeof(SA_TYPE *), hipMemcpyDeviceToHost);
	manageCudaError();

#if defined FM_COMP_32 || FM_COMP_64
	FM_COMP_TYPE **count = (FM_COMP_TYPE **) malloc(strand->n_count * sizeof(FM_COMP_TYPE *)); 
	check_malloc(count, "reverse_strand_O");
	hipMemcpy(count, strand->count, strand->n_count * sizeof(FM_COMP_TYPE *), hipMemcpyDeviceToHost);
	manageCudaError();
#endif

	for (SA_TYPE i=0; i<strand->n_desp; i++) {
		hipFree(desp[i]);
#if defined FM_COMP_32 || FM_COMP_64
		hipFree(count[i]);
#endif
	}

	hipFree(strand->desp);
	if (reverse != NULL){
		hipFree(reverse->desp);	
	}
	free(desp);

#if defined FM_COMP_32 || FM_COMP_64
	hipFree(strand->count);
	if (reverse != NULL) {
		hipFree(reverse->count);

	}
	free(count);
#endif

}

//void declare_blocked_results_list_cpu(blocked_results_lists *lists, uintmax_t max_results, uintmax_t num_lists) {
//	hipHostMalloc((void**) &(lists->results), max_results * num_lists * sizeof(result));
//	hipHostMalloc((void**) &(lists->num_results), num_lists * sizeof(uintmax_t));
//}
//
//void declare_blocked_results_list_gpu(blocked_results_lists *lists, uintmax_t max_results, uintmax_t num_lists) {
//	hipMalloc((void**) &(lists->results), max_results * num_lists * sizeof(result));
//	hipMalloc((void**) &(lists->num_results), num_lists * sizeof(uintmax_t));
//}
//
//void copy_blocked_results_list_gpu(blocked_results_lists *lists_gpu, blocked_results_lists *lists_cpu, uintmax_t max_results, uintmax_t num_lists) {
//	hipMemcpy(lists_gpu->results, lists_cpu->results, max_results * num_lists * sizeof(result), hipMemcpyHostToDevice);
//	hipMemcpy(lists_gpu->num_results, lists_cpu->num_results, num_lists * sizeof(uintmax_t), hipMemcpyHostToDevice);
//}
//
//void copy_blocked_results_list_cpu(blocked_results_lists *lists_cpu, blocked_results_lists *lists_gpu, uintmax_t max_results, uintmax_t num_lists) {
//	hipMemcpy(lists_cpu->results, lists_gpu->results, max_results * num_lists * sizeof(result), hipMemcpyDeviceToHost);
//	hipMemcpy(lists_cpu->num_results, lists_gpu->num_results, num_lists * sizeof(uintmax_t), hipMemcpyDeviceToHost);
//}
//
//void write_blocked_results(blocked_results_lists *r_list, exome* ex, comp_vector *S, comp_vector *Si, vector *C, comp_matrix *O, comp_matrix *Oi, char *mappings, uintmax_t nW, bool type, FILE *fp, uintmax_t max_results, uintmax_t num_lists, uintmax_t block_read_index) {
//
//	result *r;
//	bool found;
//
//	char search[MAXLINE+1];
//
//	for (uintmax_t k=0; k<num_lists; k++) {
//
//		found = false;
//
//		search[0] = '\0';
//		strncat(search, mappings + k*MAXLINE, nW);
//
//		for (uintmax_t i=0; i<r_list->num_results[k]; i++) {
//			r = &r_list->results[k*max_results + i];
//			manage_single_result(r, ex, S, Si, C, O, Oi, search, type, fp, block_read_index + k, &found);
//		}
//
//	}
//
//}
//

__global__ void BWExactSearchBackwardGPU(uint8_t *W, uint64_t *nW, intmax_t *k, intmax_t *l, intmax_t k_ini, intmax_t l_ini, SA_TYPE *C, SA_TYPE *C1, comp_matrix O) {

	intmax_t i;
	intmax_t k2, l2;
	uintmax_t offset  = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ SA_TYPE Cshared[4];
	__shared__ SA_TYPE C1shared[4];

	if (threadIdx.x<4) {
		Cshared[threadIdx.x]  = C[threadIdx.x];
		C1shared[threadIdx.x] = C1[threadIdx.x];
	}

	__syncthreads();

	k2 = k_ini; l2 = l_ini;

	for (i=nW[offset]-1; (k2<=l2) && (i>=0); i--)
		BWiterationGPU(k2, l2, k2, l2, W[offset*MAXLINE+i], Cshared, C1shared, O);

	k[offset] = k2;
	l[offset] = l2;

}

__global__ void BWExactSearchForwardGPU(uint8_t *W, uint64_t *nW, intmax_t *k, intmax_t *l, intmax_t k_ini, intmax_t l_ini, SA_TYPE *C, SA_TYPE *C1, comp_matrix O) {

	intmax_t i;
	intmax_t k2, l2;
	uintmax_t offset  = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ SA_TYPE Cshared[4];
	__shared__ SA_TYPE C1shared[4];

	if (threadIdx.x<4) {
		Cshared[threadIdx.x] = C[threadIdx.x];
		C1shared[threadIdx.x] = C1[threadIdx.x];
	}

	__syncthreads();

	k2 = k_ini;  l2 = l_ini;

	for (i=0; (k2<=l2) && (i<nW[offset]); i++)
		BWiterationGPU(k2, l2, k2, l2, W[offset*MAXLINE+i], Cshared, C1shared, O);

	k[offset] = k2;
	l[offset] = l2;

}

__global__ void BWExactSearchBackwardVectorGPU(uint8_t *W, uint64_t *nW, intmax_t *k, intmax_t *l, intmax_t k_ini, intmax_t l_ini, SA_TYPE *C, SA_TYPE *C1, comp_matrix O) {

	intmax_t i;
	intmax_t k2, l2;
	uintmax_t offset  = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ SA_TYPE Cshared[4];
	__shared__ SA_TYPE C1shared[4];

	if (threadIdx.x<4) {
		Cshared[threadIdx.x] = C[threadIdx.x];
		C1shared[threadIdx.x] = C1[threadIdx.x];
	}

	__syncthreads();

	k2 = k_ini;  l2 = l_ini;

	for (i=nW[offset]-1; i>=0; i--) {

		BWiterationGPU(k2, l2, k2, l2, W[offset*MAXLINE + i], Cshared, C1shared, O);

		k[offset*MAXLINE+i] = k2;
		l[offset*MAXLINE+i] = l2;

		if (k2 > l2) {
			i--;
			break;
		}

	}

	for(;i>=0; i--) {
		k[offset*MAXLINE+i] = k2;
		l[offset*MAXLINE+i] = l2;
	}

}

__global__ void BWExactSearchForwardVectorGPU(uint8_t *W, uint64_t *nW, intmax_t *k, intmax_t *l, intmax_t k_ini, intmax_t l_ini, SA_TYPE *C, SA_TYPE *C1, comp_matrix O) {

	intmax_t i;
	intmax_t k2, l2;
	uintmax_t offset  = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ SA_TYPE Cshared[4];
	__shared__ SA_TYPE C1shared[4];

	if (threadIdx.x<4) {
		Cshared[threadIdx.x] = C[threadIdx.x];
		C1shared[threadIdx.x] = C1[threadIdx.x];
	}

	__syncthreads();

	k2 = k_ini;  l2 = l_ini;

	for (i=0; i<nW[offset]; i++) {

		BWiterationGPU(k2, l2, k2, l2, W[offset*MAXLINE + i], Cshared, C1shared, O);

		k[offset*MAXLINE+i] = k2;
		l[offset*MAXLINE+i] = l2;

	}

}

//SOME TESTS:
/* __global__ void BWExactFinalResultsBackwardGPU(uint8_t *W, SA_TYPE *C, SA_TYPE *C1, comp_matrix O, results_list rl_prev, results_list rl_next, SA_TYPE chunk_size, SA_TYPE *stack_size) { */
/*   SA_TYPE k, l; */
/*   int16_t start, pos, pos_start, end; */
/*   unsigned read_index, read_offset; */
/*   SA_TYPE offset  = blockIdx.x * blockDim.x + threadIdx.x; */
/*   result *r_prev, *r_next; */

/*   __shared__ SA_TYPE Cshared[4]; */
/*   __shared__ SA_TYPE C1shared[4]; */

/*   if (offset==0) */
/*     *stack_size=0; */

/*   if (threadIdx.x<4) { */
/*     Cshared[threadIdx.x] = C[threadIdx.x]; */
/*     C1shared[threadIdx.x] = C1[threadIdx.x]; */
/*   } */

/*   __syncthreads(); */

/*   r_prev = &rl_prev.list[offset]; */

/*   start      = r_prev->start; */
/*   pos        = r_prev->pos; */
/*   end        = r_prev->end; */
/*   k          = r_prev->k; */
/*   l          = r_prev->l; */
/*   read_index = r_prev->read_index; */
/*   read_offset = /\*read_index*\/offset*MAXLINE; */

/*   pos_start = pos - chunk_size; */
/*   if (pos_start < start) pos_start = start; */

/*   for(; pos>=pos_start; pos--) { */
/*     BWiterationGPU(k, l, k, l, W[read_offset + pos], Cshared, C1shared, O); */
/*     if (k > l) { */
/*       pos=start-1; break; */
/*     } */
/*   } */

/*   r_next = &rl_next.list[/\*atomicAdd(stack_size,1)*\/offset]; */

/*   r_next->start = start; */
/*   r_next->pos = pos; */
/*   r_next->end = end; */
/*   r_next->k = k; */
/*   r_next->l = l; */
/*   r_next->read_index = read_index; */

/* } */

///////////////////////////////////////////////////MULTI-ERROR///////////////////////////////////////////////////////////
//__global__ void init_listsGPU(blocked_results_lists rl_prev, blocked_results_lists rl_next, blocked_results_lists rl_prev_i, blocked_results_lists rl_next_i, blocked_results_lists rl_final) {
//
//	SA_TYPE offset  = blockIdx.x * blockDim.x + threadIdx.x;
//
//	rl_prev.num_results[offset]   = 0;
//	rl_next.num_results[offset]   = 0;
//	rl_prev_i.num_results[offset] = 0;
//	rl_next_i.num_results[offset] = 0;
//	rl_final.num_results[offset]  = 0;
//
//}
//
//__global__ void BWExactSearchBackwardBlockedGPU(uint8_t *W, SA_TYPE nW, SA_TYPE *C, SA_TYPE *C1, comp_matrix O, result r, blocked_results_lists rl_prev, uintmax_t max_results) {
//
//	SA_TYPE offset  = blockIdx.x * blockDim.x + threadIdx.x;
//
//	SA_TYPE k2, l2;
//	result *r_iterator;
//	intmax_t i;
//
//	__shared__ SA_TYPE Cshared[4];
//	__shared__ SA_TYPE C1shared[4];
//
//	if (threadIdx.x<4) {
//		Cshared[threadIdx.x] = C[threadIdx.x];
//		C1shared[threadIdx.x] = C1[threadIdx.x];
//	}
//
//	__syncthreads();
//
//	k2 = r.k;
//	l2 = r.l;
//
//	for (i=r.pos; (k2<=l2) && (i>=r.start); i--)
//		BWiterationGPU(k2, l2, k2, l2, W[offset*MAXLINE+i], Cshared, C1shared, O);
//
//	if (k2 <= l2) {
//		r_iterator = rl_prev.results + offset * max_results;
//
//		init_result(r_iterator, 0);
//		bound_result(r_iterator, 0, r.end);
//		change_result(r_iterator, k2, l2, i);
//
//		rl_prev.num_results[offset] = 1; //Init the number of results
//	} else {
//		rl_prev.num_results[offset] = 0; //Init the number of results
//	}
//
//}
//
//__global__ void BWExactSearchForwardBlockedGPU(uint8_t *W, SA_TYPE nW, SA_TYPE *C, SA_TYPE *C1, comp_matrix O, result r, blocked_results_lists rl_prev, uintmax_t max_results) {
//
//	SA_TYPE offset  = blockIdx.x * blockDim.x + threadIdx.x;
//
//	SA_TYPE k2, l2;
//	result *r_iterator;
//	intmax_t i;
//
//	__shared__ SA_TYPE Cshared[4];
//	__shared__ SA_TYPE C1shared[4];
//
//	if (threadIdx.x<4) {
//		Cshared[threadIdx.x] = C[threadIdx.x];
//		C1shared[threadIdx.x] = C1[threadIdx.x];
//	}
//
//	__syncthreads();
//
//	k2 = r.k;
//	l2 = r.l;
//
//	for (i=r.pos; (k2<=l2) && (i<=r.end); i++)
//		BWiterationGPU(k2, l2, k2, l2, W[offset*MAXLINE+i], Cshared, C1shared, O);
//
//	if (k2 <= l2) {
//		r_iterator = rl_prev.results + offset * max_results;
//
//		init_result(r_iterator, 1);
//		bound_result(r_iterator, r.start, nW-1);
//		change_result(r_iterator, k2, l2, i);
//
//		rl_prev.num_results[offset] = 1; //Init the number of results
//	} else {
//		rl_prev.num_results[offset] = 0; //Init the number of results
//	}
//
//}
//
//__global__ void BWExactFinalResultsBackwardBlockedGPU(uint8_t *W, SA_TYPE *C, SA_TYPE *C1, comp_matrix O, blocked_results_lists rl_prev, blocked_results_lists rl_next_i, int16_t block_size, int16_t last_block, uintmax_t max_results) {
//
//	SA_TYPE offset  = blockIdx.x * blockDim.x + threadIdx.x;
//	result *rl_next_i_results = rl_next_i.results + offset * max_results;
//	SA_TYPE rl_next_i_num_results = rl_next_i.num_results[offset];
//
//	SA_TYPE k, l;
//	int16_t start, pos, current_block;
//
//	__shared__ SA_TYPE Cshared[4];
//	__shared__ SA_TYPE C1shared[4];
//
//	if (threadIdx.x<4) {
//		Cshared[threadIdx.x] = C[threadIdx.x];
//		C1shared[threadIdx.x] = C1[threadIdx.x];
//	}
//
//	result *r_iterator;
//
//	for (uintmax_t ii=0; ii < rl_prev.num_results[offset]; ii++) {
//
//		r_iterator = rl_prev.results + offset * max_results + ii;
//
//		start = r_iterator->start;
//		pos   = r_iterator->pos;
//
//		k = r_iterator->k;
//		l = r_iterator->l;
//
//		current_block = pos / block_size;
//
//		if ((current_block < last_block) || (pos == start-1)) { // Current block will be always >= start and previous results are propagated
//
//		} else {
//
//			if (current_block > last_block) { //Not in last previsited block
//
//				continue;
//
//			} else { //I am in the last previsited block
//
//				if ((pos + 1) % block_size) { //I am not in the first element of the block
//				} else { //I am in the first element in the block (all the block must be processed)
//					continue;
//				}
//
//			}
//
//		}
//
//		__syncthreads();
//
//		for(int16_t i=pos; i>=start; i--) {
//			BWiterationGPU(k, l, k, l, W[offset*MAXLINE+i], Cshared, C1shared, O);
//			if (k > l) break;
//		}
//
//		if (k <= l) {
//			change_result(r_iterator, k, l, start-1);
//			add_resultGPU(r_iterator, rl_next_i_results, rl_next_i_num_results, max_results);
//		}
//
//	} //r_prev
//
//	rl_next_i.num_results[offset] = rl_next_i_num_results;
//
//}
//
//__global__ void BWExactFinalResultsForwardBlockedGPU(uint8_t *W, SA_TYPE *C, SA_TYPE *C1, comp_matrix O, blocked_results_lists rl_prev, blocked_results_lists rl_next_i, int16_t block_size, int16_t last_block, uintmax_t max_results) {
//
//	SA_TYPE offset  = blockIdx.x * blockDim.x + threadIdx.x;
//	result *rl_next_i_results = rl_next_i.results + offset * max_results;
//	SA_TYPE rl_next_i_num_results = rl_next_i.num_results[offset];
//
//	SA_TYPE k, l;
//	int16_t pos, end;
//	int16_t current_block;
//
//	__shared__ SA_TYPE Cshared[4];
//	__shared__ SA_TYPE C1shared[4];
//
//	if (threadIdx.x<4) {
//		Cshared[threadIdx.x] = C[threadIdx.x];
//		C1shared[threadIdx.x] = C1[threadIdx.x];
//	}
//
//	result *r_iterator;
//
//	for (uintmax_t ii=0; ii < rl_prev.num_results[offset]; ii++) {
//
//		r_iterator = rl_prev.results + offset * max_results + ii;
//
//		pos   = r_iterator->pos;
//		end   = r_iterator->end;
//
//		k = r_iterator->k;
//		l = r_iterator->l;
//
//		current_block = pos / block_size;
//
//		if ( (current_block > last_block) || (pos == end+1) ) { // Current block will be always <= end and previous results are propagated
//
//		} else {
//
//			if (current_block < last_block) { //Not in last previsited block
//
//				continue;
//
//			} else { //I am in the last previsited block
//
//				if (pos % block_size) { //I am not in the first element of the block
//				} else { //I am in the first element in the block (all the block must be processed)
//					continue;
//				}
//
//			}
//
//		}
//
//		__syncthreads();
//
//		for(int16_t i=pos; i<=end; i++) {
//			BWiterationGPU(k, l, k, l, W[offset*MAXLINE+i], Cshared, C1shared, O);
//			if (k > l) break;
//		}
//
//		if (k <= l) {
//			change_result(r_iterator, k, l, end+1);
//			add_resultGPU(r_iterator, rl_next_i_results, rl_next_i_num_results, max_results);
//		}
//
//	} //r_prev
//
//	rl_next_i.num_results[offset] = rl_next_i_num_results;
//
//}
//
//__device__ uintmax_t BWExactFinalResultForwardGPU(uint8_t *W, SA_TYPE *C, SA_TYPE *C1, comp_matrix *O, result *r_iterator, result *rl_next_i_results, uintmax_t rl_next_i_num_results, int16_t block_size, int16_t last_block, uintmax_t max_results) {
//
//	SA_TYPE k, l;
//	int16_t pos, end;
//	int16_t current_block;
//
//	pos   = r_iterator->pos;
//	end   = r_iterator->end;
//
//	k = r_iterator->k;
//	l = r_iterator->l;
//
//	current_block = pos / block_size;
//
//	if ( (current_block > last_block) || (pos == end+1) ) { // Current block will be always <= end and previous results are propagated
//
//	} else {
//
//		if (current_block < last_block) { //Not in last previsited block
//
//			return rl_next_i_num_results;
//
//		} else { //I am in the last previsited block
//
//			if (pos % block_size) { //I am not in the first element of the block
//			} else { //I am in the first element in the block (all the block must be processed)
//				return rl_next_i_num_results;
//			}
//
//		}
//
//	}
//
//	for(int16_t i=pos; i<=end; i++) {
//		BWiterationGPU(k, l, k, l, W[i], C, C1, *O);
//		if (k > l) break;
//	}
//
//	if (k <= l) {
//		change_result(r_iterator, k, l, end+1);
//		add_resultGPU(r_iterator, rl_next_i_results, rl_next_i_num_results, max_results);
//	}
//
//	return rl_next_i_num_results;
//
//}
//
//__global__ void BWBranchFinalResultsForwardBlockedGPU(uint8_t *W, SA_TYPE *C, SA_TYPE *C1, comp_matrix O, blocked_results_lists rl_prev, blocked_results_lists rl_next, int16_t block_size, int16_t last_block, intmax_t max_results, uint8_t nA) {
//
//	SA_TYPE offset  = blockIdx.x * blockDim.x + threadIdx.x;
//	result *rl_next_results = rl_next.results + offset * max_results;
//	uintmax_t rl_next_num_results = rl_next.num_results[offset];
//
//	SA_TYPE k, l, k_aux, l_aux;
//	int16_t end, pos;
//	int16_t r_num_mismatches;
//	bool no_previous;
//	int16_t last_err_pos;
//	uint8_t last_err_kind;
//	uint8_t last_err_base;
//
//	result *r_iterator;
//
//	for (uintmax_t ii=0; ii < rl_prev.num_results[offset]; ii++) {
//
//		r_iterator = rl_prev.results + offset * max_results + ii;
//
//		if (r_iterator->dir != 1) continue;
//
//		end = r_iterator->end;
//		pos = r_iterator->pos;
//
//		if (pos > end) {
//			add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//			continue;
//		}
//
//		no_previous = true;
//		r_num_mismatches = r_iterator->num_mismatches-1;
//		if (r_num_mismatches>-1) {
//			last_err_pos  = r_iterator->err_pos[r_num_mismatches];
//			last_err_kind = r_iterator->err_kind[r_num_mismatches];
//			last_err_base = r_iterator->err_base[r_num_mismatches];
//		} else {
//			last_err_pos  = -10;
//			last_err_kind = 0;
//			last_err_base = (uint8_t) -1;
//		}
//
//		k = r_iterator->k;
//		l = r_iterator->l;
//
//		add_mismatch(r_iterator, DELETION, (uint8_t) -1, pos);
//
//		if (last_err_pos == pos - 1) { //Previous MISMATCH or DELETION
//
//			if (last_err_kind == MISMATCH) { //Previous MISMATCH
//
//				//Deletion
//				if (W[offset*MAXLINE+pos]!=last_err_base) {
//					change_result(r_iterator, k, l, pos+1);
//					rl_next_num_results = BWExactFinalResultForwardGPU(W, C, C1, &O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//				}
//
//				for (uint8_t b=0;b<nA;b++) {
//
//					BWiterationGPU(k, l, k_aux, l_aux, b, C, C1, O);
//
//					if (k_aux > l_aux) continue;
//
//					//Insertion
//					if (b!=W[offset*MAXLINE+last_err_pos]) {
//						change_result(r_iterator, k_aux, l_aux, pos);
//						modify_last_mismatch2(r_iterator, INSERTION, b);
//						rl_next_num_results = BWExactFinalResultForwardGPU(W, C, C1, &O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//					}
//
//					//Mismatch
//					if (b!=W[offset*MAXLINE+pos]) {
//						change_result(r_iterator, k_aux, l_aux, pos+1);
//						modify_last_mismatch2(r_iterator, MISMATCH, b);
//						rl_next_num_results = BWExactFinalResultForwardGPU(W, C, C1, &O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//					}
//
//				}
//
//				no_previous = false;
//
//			} else if (last_err_kind == DELETION) { //Previous DELETION
//
//				//Deletion
//				change_result(r_iterator, k, l, pos+1);
//				rl_next_num_results = BWExactFinalResultForwardGPU(W, C, C1, &O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//
//				for (uint8_t b=0;b<nA;b++) {
//
//					BWiterationGPU(k, l, k_aux, l_aux, b, C, C1, O);
//
//					if (k_aux > l_aux) continue;
//
//					// NO INSERTION
//
//					//Mismatch
//					if (b!=W[offset*MAXLINE+pos]) {
//
//						if (b!=W[offset*MAXLINE+last_err_pos]) {
//							change_result(r_iterator, k_aux, l_aux, pos+1);
//							modify_last_mismatch2(r_iterator, MISMATCH, b);
//							rl_next_num_results = BWExactFinalResultForwardGPU(W, C, C1, &O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//						}
//
//					}
//
//				}
//
//				no_previous = false;
//
//			}
//
//		} else if (last_err_pos == pos) { //Previous INSERTION
//
//			//NO DELETION
//
//			for (uint8_t b=0;b<nA;b++) {
//
//				BWiterationGPU(k, l, k_aux, l_aux, b, C, C1, O);
//
//				if (k_aux > l_aux) continue;
//
//				//Insertion
//				change_result(r_iterator, k_aux, l_aux, pos);
//				modify_last_mismatch2(r_iterator, INSERTION, b);
//				rl_next_num_results = BWExactFinalResultForwardGPU(W, C, C1, &O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//
//				//Mismatch
//				if (b!=W[offset*MAXLINE+pos]) {
//
//					if (W[offset*MAXLINE+pos]!=last_err_base) {
//						r_iterator->pos = pos+1;
//						modify_last_mismatch1(r_iterator, MISMATCH);
//						rl_next_num_results = BWExactFinalResultForwardGPU(W, C, C1, &O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//					}
//
//				}
//
//			}
//
//			no_previous = false;
//
//		}
//
//		if (no_previous) { //Previous MATCH
//
//			//Deletion
//			change_result(r_iterator, k, l, pos+1);
//			rl_next_num_results = BWExactFinalResultForwardGPU(W, C, C1, &O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//
//			for (uint8_t b=0;b<nA;b++) {
//
//				BWiterationGPU(k, l, k_aux, l_aux, b, C, C1, O);
//
//				if (k_aux > l_aux) continue;
//
//				//Insertion
//				change_result(r_iterator, k_aux, l_aux, pos);
//				modify_last_mismatch2(r_iterator, INSERTION, b);
//				rl_next_num_results = BWExactFinalResultForwardGPU(W, C, C1, &O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//
//				if (b!=W[offset*MAXLINE+pos]) { //Mismatch
//					r_iterator->pos = pos+1;
//					modify_last_mismatch1(r_iterator, MISMATCH);
//					rl_next_num_results = BWExactFinalResultForwardGPU(W, C, C1, &O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//				}
//
//			}
//
//		}
//
//	}
//
//	rl_prev.num_results[offset] = 0;
//	rl_next.num_results[offset] = rl_next_num_results;
//
//}
//
//uintmax_t BWExactFinalResultForwardCPU(uint8_t *W, vector *C, vector *C1, comp_matrix *O, result *r_iterator, result *rl_next_i_results, uintmax_t rl_next_i_num_results, int16_t block_size, int16_t last_block, uintmax_t max_results) {
//
//	SA_TYPE k, l;
//	int16_t pos, end;
//	int16_t current_block;
//
//	pos   = r_iterator->pos;
//	end   = r_iterator->end;
//
//	k = r_iterator->k;
//	l = r_iterator->l;
//
//	current_block = pos / block_size;
//
//	if ( (current_block > last_block) || (pos == end+1) ) { // Current block will be always <= end and previous results are propagated
//
//	} else {
//
//		if (current_block < last_block) { //Not in last previsited block
//
//			return rl_next_i_num_results;
//
//		} else { //I am in the last previsited block
//
//			if (pos % block_size) { //I am not in the first element of the block
//			} else { //I am in the first element in the block (all the block must be processed)
//				return rl_next_i_num_results;
//			}
//
//		}
//
//	}
//
//	for(int16_t i=pos; i<=end; i++) {
//		BWiteration(k, l, k, l, W[i], C, C1, O);
//		if (k > l) break;
//	}
//
//	if (k <= l) {
//		change_result(r_iterator, k, l, end+1);
//		add_resultGPU(r_iterator, rl_next_i_results, rl_next_i_num_results, max_results);
//	}
//
//	return rl_next_i_num_results;
//
//}
//
//void BWBranchFinalResultsForwardBlockedCPU(uintmax_t num_bloques, uintmax_t tam_bloques, uint8_t *W, vector *C, vector *C1, comp_matrix *O, blocked_results_lists rl_prev, blocked_results_lists rl_next, int16_t block_size, int16_t last_block, uintmax_t max_results, uint8_t nA) {
//
//	//#pragma omp parallel for
//	for(SA_TYPE offset=0; offset < num_bloques*tam_bloques; offset++) {
//
//		result *rl_next_results = rl_next.results + offset * max_results;
//		uintmax_t rl_next_num_results = rl_next.num_results[offset];
//
//		SA_TYPE k, l, k_aux, l_aux;
//		int16_t end, pos;
//		int16_t r_num_mismatches;
//		bool no_previous;
//		int16_t last_err_pos;
//		uint8_t last_err_kind;
//		uint8_t last_err_base;
//
//		result *r_iterator;
//
//		for (uintmax_t ii=0; ii < rl_prev.num_results[offset]; ii++) {
//
//			r_iterator = rl_prev.results + offset * max_results + ii;
//
//			if (r_iterator->dir != 1) continue;
//
//			end = r_iterator->end;
//			pos = r_iterator->pos;
//
//			if (pos > end) {
//				add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//				continue;
//			}
//
//			no_previous = true;
//			r_num_mismatches = r_iterator->num_mismatches-1;
//			if (r_num_mismatches>-1) {
//				last_err_pos  = r_iterator->err_pos[r_num_mismatches];
//				last_err_kind = r_iterator->err_kind[r_num_mismatches];
//				last_err_base = r_iterator->err_base[r_num_mismatches];
//			} else {
//				last_err_pos  = -10;
//				last_err_kind = 0;
//				last_err_base = (uint8_t) -1;
//			}
//
//			k = r_iterator->k;
//			l = r_iterator->l;
//
//			add_mismatch(r_iterator, DELETION, (uint8_t) -1, pos);
//
//			if (last_err_pos == pos - 1) { //Previous MISMATCH or DELETION
//
//				if (last_err_kind == MISMATCH) { //Previous MISMATCH
//
//					//Deletion
//					if (W[offset*MAXLINE+pos]!=last_err_base) {
//						change_result(r_iterator, k, l, pos+1);
//						rl_next_num_results = BWExactFinalResultForwardCPU(W, C, C1, O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//					}
//
//					for (uint8_t b=0;b<nA;b++) {
//
//						BWiteration(k, l, k_aux, l_aux, b, C, C1, O);
//
//						if (k_aux > l_aux) continue;
//
//						//Insertion
//						if (b!=W[offset*MAXLINE+last_err_pos]) {
//							change_result(r_iterator, k_aux, l_aux, pos);
//							modify_last_mismatch2(r_iterator, INSERTION, b);
//							rl_next_num_results = BWExactFinalResultForwardCPU(W, C, C1, O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//						}
//
//						//Mismatch
//						if (b!=W[offset*MAXLINE+pos]) {
//							change_result(r_iterator, k_aux, l_aux, pos+1);
//							modify_last_mismatch2(r_iterator, MISMATCH, b);
//							rl_next_num_results = BWExactFinalResultForwardCPU(W, C, C1, O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//						}
//
//					}
//
//					no_previous = false;
//
//				} else if (last_err_kind == DELETION) { //Previous DELETION
//
//					//Deletion
//					change_result(r_iterator, k, l, pos+1);
//					rl_next_num_results = BWExactFinalResultForwardCPU(W, C, C1, O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//
//					for (uint8_t b=0;b<nA;b++) {
//
//						BWiteration(k, l, k_aux, l_aux, b, C, C1, O);
//
//						if (k_aux > l_aux) continue;
//
//						// NO INSERTION
//
//						//Mismatch
//						if (b!=W[offset*MAXLINE+pos]) {
//
//							if (b!=W[offset*MAXLINE+last_err_pos]) {
//								change_result(r_iterator, k_aux, l_aux, pos+1);
//								modify_last_mismatch2(r_iterator, MISMATCH, b);
//								rl_next_num_results = BWExactFinalResultForwardCPU(W, C, C1, O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//							}
//
//						}
//
//					}
//
//					no_previous = false;
//
//				}
//
//			} else if (last_err_pos == pos) { //Previous INSERTION
//
//				//NO DELETION
//
//				for (uint8_t b=0;b<nA;b++) {
//
//					BWiteration(k, l, k_aux, l_aux, b, C, C1, O);
//
//					if (k_aux > l_aux) continue;
//
//					//Insertion
//					change_result(r_iterator, k_aux, l_aux, pos);
//					modify_last_mismatch2(r_iterator, INSERTION, b);
//					rl_next_num_results = BWExactFinalResultForwardCPU(W, C, C1, O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//
//					//Mismatch
//					if (b!=W[offset*MAXLINE+pos]) {
//
//						if (W[offset*MAXLINE+pos]!=last_err_base) {
//							r_iterator->pos = pos+1;
//							modify_last_mismatch1(r_iterator, MISMATCH);
//							rl_next_num_results = BWExactFinalResultForwardCPU(W, C, C1, O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//						}
//
//					}
//
//				}
//
//				no_previous = false;
//
//			}
//
//			if (no_previous) { //Previous MATCH
//
//				//Deletion
//				change_result(r_iterator, k, l, pos+1);
//				rl_next_num_results = BWExactFinalResultForwardCPU(W, C, C1, O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//
//				for (uint8_t b=0;b<nA;b++) {
//
//					BWiteration(k, l, k_aux, l_aux, b, C, C1, O);
//
//					if (k_aux > l_aux) continue;
//
//					//Insertion
//					change_result(r_iterator, k_aux, l_aux, pos);
//					modify_last_mismatch2(r_iterator, INSERTION, b);
//					rl_next_num_results = BWExactFinalResultForwardCPU(W, C, C1, O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//
//					if (b!=W[offset*MAXLINE+pos]) { //Mismatch
//						r_iterator->pos = pos+1;
//						modify_last_mismatch1(r_iterator, MISMATCH);
//						rl_next_num_results = BWExactFinalResultForwardCPU(W, C, C1, O, r_iterator, rl_next_results, rl_next_num_results, block_size, last_block, max_results);
//					}
//
//				}
//
//			}
//
//		}
//
//		rl_prev.num_results[offset] = 0;
//		rl_next.num_results[offset] = rl_next_num_results;
//
//	}
//
//}
//
//__global__ void BWExactPartialResultsBackwardBlockedGPU(uint8_t *W, SA_TYPE *C, SA_TYPE *C1, comp_matrix O, blocked_results_lists rl_prev, blocked_results_lists rl_next, blocked_results_lists rl_next_i, int16_t block_size, int16_t last_block, uintmax_t max_results) {
//
//	SA_TYPE offset  = blockIdx.x * blockDim.x + threadIdx.x;
//	result *rl_next_results = rl_next.results + offset * max_results;
//	uintmax_t rl_next_num_results = 0;
//	result *rl_next_i_results = rl_next_i.results + offset * max_results;
//	uintmax_t rl_next_i_num_results = rl_next_i.num_results[offset];
//
//	SA_TYPE k, l, k_next, l_next;
//	int16_t start, pos;
//	int16_t current_block, last_block_pos;
//	bool complete_search;
//
//	__shared__ SA_TYPE Cshared[4];
//	__shared__ SA_TYPE C1shared[4];
//
//	if (threadIdx.x<4) {
//		Cshared[threadIdx.x] = C[threadIdx.x];
//		C1shared[threadIdx.x] = C1[threadIdx.x];
//	}
//
//	result *r_iterator;
//	SA_TYPE results, results_next;
//
//	for (uintmax_t ii=0; ii < rl_prev.num_results[offset]; ii++) {
//
//		r_iterator = rl_prev.results + offset * max_results + ii;
//
//		start  = r_iterator->start;
//		pos    = r_iterator->pos;
//
//		k_next = r_iterator->k;
//		l_next = r_iterator->l;
//		results_next = l_next - k_next;
//
//		current_block = pos / block_size;
//
//		if ((current_block < last_block) || (pos == start-1)) { // Current block will be always >= start and previous results are propagated
//
//			last_block_pos = start;
//			complete_search = true;
//
//		} else {
//
//			if (current_block > last_block) { //Not in last previsited block
//
//				if ((pos + 1) % block_size) { //Not in first element of the block
//					last_block_pos = (current_block-1) * block_size;
//				} else { //I am in the first element in the block (all the block must be processed)
//					last_block_pos = current_block * block_size;
//				}
//
//				complete_search = false;
//
//			} else { //I am in the last previsited block
//
//				if ((pos + 1) % block_size) { //I am not in the first element of the block
//					last_block_pos = start;
//					complete_search = true;
//				} else { //I am in the first element of the block (all the block must be processed)
//					last_block_pos = current_block * block_size;
//					complete_search = false;
//				}
//
//			}
//
//		}
//
//		__syncthreads();
//
//		for(int16_t i=pos; i>=last_block_pos; i--) {
//
//			k = k_next;
//			l = l_next;
//
//			if (k > l) break;
//
//			BWiterationGPU(k, l, k_next, l_next, W[offset*MAXLINE+i], Cshared, C1shared, O);
//			results      = results_next;
//			results_next = l_next - k_next;
//			if (results == results_next) continue;
//
//			change_result(r_iterator, k, l, i);
//			add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//		}
//
//		if (complete_search && k_next <= l_next) {
//			change_result(r_iterator, k_next, l_next, start-1);
//			add_resultGPU(r_iterator, rl_next_i_results, rl_next_i_num_results, max_results);
//		}
//
//	} //r_prev
//
//	rl_next.num_results[offset] = rl_next_num_results;
//	rl_next_i.num_results[offset] = rl_next_i_num_results;
//
//}
//
//__global__ void BWExactPartialResultsForwardBlockedGPU(uint8_t *W, SA_TYPE *C, SA_TYPE *C1, comp_matrix O, blocked_results_lists rl_prev, blocked_results_lists rl_next, blocked_results_lists rl_next_i, int16_t block_size, int16_t last_block, uintmax_t max_results) {
//
//	SA_TYPE offset  = blockIdx.x * blockDim.x + threadIdx.x;
//	result *rl_next_results = rl_next.results + offset * max_results;
//	uintmax_t rl_next_num_results = 0;
//	result *rl_next_i_results = rl_next_i.results + offset * max_results;
//	uintmax_t rl_next_i_num_results = rl_next_i.num_results[offset];
//
//	SA_TYPE k, l, k_next, l_next;
//	int16_t pos, end;
//	int16_t current_block, last_block_pos;
//	SA_TYPE results, results_next;
//	bool complete_search;
//
//	__shared__ SA_TYPE Cshared[4];
//	__shared__ SA_TYPE C1shared[4];
//
//	if (threadIdx.x<4) {
//		Cshared[threadIdx.x] = C[threadIdx.x];
//		C1shared[threadIdx.x] = C1[threadIdx.x];
//	}
//
//	result *r_iterator;
//
//
//	for (uintmax_t ii=0; ii < rl_prev.num_results[offset]; ii++) {
//
//		r_iterator = rl_prev.results + offset * max_results + ii;
//
//		pos   = r_iterator->pos;
//		end   = r_iterator->end;
//
//		k_next = r_iterator->k;
//		l_next = r_iterator->l;
//		results_next = l_next - k_next;
//
//		current_block = pos / block_size;
//
//		if ( (current_block > last_block) || (pos == end+1) ) { // Current block will be always <= end and previous results are propagated
//
//			last_block_pos = end;
//			complete_search = true;
//
//		} else {
//
//			if (current_block < last_block) { //Not in last previsited block
//
//				if (pos % block_size) { //Not in first element of the block
//					last_block_pos = (current_block+2) * block_size - 1;
//				} else { //I am in the first element in the block (all the block must be processed)
//					last_block_pos = (current_block+1) * block_size - 1;
//				}
//
//				complete_search = false;
//
//			} else { //I am in the last previsited block
//
//				if (pos % block_size) { //I am not in the first element of the block
//					last_block_pos = end;
//					complete_search = true;
//				} else { //I am in the first element in the block (all the block must be processed)
//					last_block_pos = (current_block+1) * block_size - 1;
//					complete_search = false;
//				}
//
//			}
//
//		}
//
//		__syncthreads();
//
//		for(int16_t i=pos; i<=last_block_pos; i++) {
//
//			k = k_next;
//			l = l_next;
//
//			if (k > l) break;
//
//			BWiterationGPU(k, l, k_next, l_next, W[offset*MAXLINE+i], Cshared, C1shared, O);
//			results      = results_next;
//			results_next = l_next - k_next;
//			if (results == results_next) continue;
//
//			change_result(r_iterator, k, l, i);
//			add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//		}
//
//		if (complete_search && k_next <= l_next) {
//			change_result(r_iterator, k_next, l_next, end+1);
//			add_resultGPU(r_iterator, rl_next_i_results, rl_next_i_num_results, max_results);
//		}
//
//	} //r_prev
//
//	rl_next.num_results[offset] = rl_next_num_results;
//	rl_next_i.num_results[offset] = rl_next_i_num_results;
//
//}
//
//__global__ void BWBranchPartialResultsBackwardBlockedGPU(uint8_t *W, SA_TYPE *C, SA_TYPE *C1, comp_matrix O, blocked_results_lists rl_prev, blocked_results_lists rl_next, uintmax_t max_results, uint8_t nA) {
//
//	SA_TYPE offset  = blockIdx.x * blockDim.x + threadIdx.x;
//	result *rl_next_results = rl_next.results + offset * max_results;
//	uintmax_t rl_next_num_results = 0;
//
//	SA_TYPE k, l, k_aux, l_aux;
//	int16_t start, pos;
//	int16_t r_num_mismatches;
//	bool no_previous;
//	int16_t last_err_pos;
//	uint8_t last_err_kind;
//	uint8_t last_err_base;
//
//	__shared__ SA_TYPE Cshared[4];
//	__shared__ SA_TYPE C1shared[4];
//
//	if (threadIdx.x<4) {
//		Cshared[threadIdx.x] = C[threadIdx.x];
//		C1shared[threadIdx.x] = C1[threadIdx.x];
//	}
//
//	result *r_iterator;
//
//	for (uintmax_t ii=0; ii < rl_prev.num_results[offset]; ii++) {
//
//		r_iterator = rl_prev.results + offset * max_results + ii;
//
//		if (r_iterator->dir != 0) continue;
//
//		start = r_iterator->start;
//		pos   = r_iterator->pos;
//
//		if (pos < start) {
//			add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//			continue;
//		}
//
//		no_previous = true;
//		r_num_mismatches = r_iterator->num_mismatches-1;
//		if (r_num_mismatches>-1) {
//			last_err_pos  = r_iterator->err_pos[r_num_mismatches];
//			last_err_kind = r_iterator->err_kind[r_num_mismatches];
//			last_err_base = r_iterator->err_base[r_num_mismatches];
//		} else {
//			last_err_pos  = -10;
//			last_err_kind = 0;
//			last_err_base = (uint8_t) -1;
//		}
//
//		k = r_iterator->k;
//		l = r_iterator->l;
//
//		add_mismatch(r_iterator, DELETION, (uint8_t) -1, pos);
//
//		__syncthreads();
//
//		if (last_err_pos == pos + 1) { //Previous MISMATCH or DELETION
//
//			if (last_err_kind == MISMATCH) { //Previous MISMATCH
//
//				//Deletion
//				if (W[offset*MAXLINE+pos]!=last_err_base) {
//					change_result(r_iterator, k, l, pos-1);
//					add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//				}
//
//				for (uint8_t b=0;b<nA;b++) {
//
//					BWiterationGPU(k, l, k_aux, l_aux, b, Cshared, C1shared, O);
//
//					if (k_aux > l_aux) continue;
//
//					//Insertion
//					if (b!=W[offset*MAXLINE+last_err_pos]) {
//						change_result(r_iterator, k_aux, l_aux, pos);
//						modify_last_mismatch2(r_iterator, INSERTION, b);
//						add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//					}
//
//					//Mismatch
//					if (b!=W[offset*MAXLINE+pos]) {
//						change_result(r_iterator, k_aux, l_aux, pos-1);
//						modify_last_mismatch2(r_iterator, MISMATCH, b);
//						add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//					}
//
//				}
//
//				no_previous = false;
//
//			} else if (last_err_kind == DELETION) { //Previous DELETION
//
//				//Deletion
//				change_result(r_iterator, k, l, pos-1);
//				add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//				for (uint8_t b=0;b<nA;b++) {
//
//					BWiterationGPU(k, l, k_aux, l_aux, b, Cshared, C1shared, O);
//
//					if (k_aux > l_aux) continue;
//
//					// NO INSERTION
//
//					if (b!=W[offset*MAXLINE+pos]) { //Mismatch
//
//						if (b!=W[offset*MAXLINE+last_err_pos]) {
//							change_result(r_iterator, k_aux, l_aux, pos-1);
//							modify_last_mismatch2(r_iterator, MISMATCH, b);
//							add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//						}
//
//					}
//
//				}
//
//				no_previous = false;
//
//			}
//
//		} else if (last_err_pos == pos) { //Previous INSERTION
//
//			//NO DELETION
//
//			for (uint8_t b=0;b<nA;b++) {
//
//				BWiterationGPU(k, l, k_aux, l_aux, b, Cshared, C1shared, O);
//
//				if (k_aux > l_aux) continue;
//
//				//Insertion
//				change_result(r_iterator, k_aux, l_aux, pos);
//				modify_last_mismatch2(r_iterator, INSERTION, b);
//				add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//				//Mismatch
//				if (b!=W[offset*MAXLINE+pos]) {
//
//					if (W[offset*MAXLINE+pos]!=last_err_base) {
//						r_iterator->pos = pos-1;
//						modify_last_mismatch1(r_iterator, MISMATCH);
//						add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//					}
//
//				}
//
//			}
//
//			no_previous = false;
//
//		}
//
//		if (no_previous) { //Previous MATCH
//
//			//Deletion
//			change_result(r_iterator, k, l, pos-1);
//			add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//			for (uint8_t b=0;b<nA;b++) {
//
//				BWiterationGPU(k, l, k_aux, l_aux, b, Cshared, C1shared, O);
//
//				if (k_aux > l_aux) continue;
//
//				//Insertion
//				change_result(r_iterator, k_aux, l_aux, pos);
//				modify_last_mismatch2(r_iterator, INSERTION, b);
//				add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//				//Mismatch
//				if (b!=W[offset*MAXLINE+pos]) {
//					r_iterator->pos = pos-1;
//					modify_last_mismatch1(r_iterator, MISMATCH);
//					add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//				}
//
//			}
//
//		}
//
//	}
//
//	rl_next.num_results[offset] = rl_next_num_results;
//
//}
//
//__global__ void BWBranchPartialResultsForwardBlockedGPU(uint8_t *W, SA_TYPE *C, SA_TYPE *C1, comp_matrix O, blocked_results_lists rl_prev, blocked_results_lists rl_next, uintmax_t max_results, uint8_t nA) {
//
//	SA_TYPE offset  = blockIdx.x * blockDim.x + threadIdx.x;
//	result *rl_next_results = rl_next.results + offset * max_results;
//	uintmax_t rl_next_num_results = 0;
//
//	SA_TYPE k, l, k_aux, l_aux;
//	int16_t end, pos;
//	int16_t r_num_mismatches;
//	bool no_previous;
//	int16_t last_err_pos;
//	uint8_t last_err_kind;
//	uint8_t last_err_base;
//
//	__shared__ SA_TYPE Cshared[4];
//	__shared__ SA_TYPE C1shared[4];
//
//	if (threadIdx.x<4) {
//		Cshared[threadIdx.x] = C[threadIdx.x];
//		C1shared[threadIdx.x] = C1[threadIdx.x];
//	}
//
//	result *r_iterator;
//
//	for (uintmax_t ii=0; ii < rl_prev.num_results[offset]; ii++) {
//
//		r_iterator = rl_prev.results + offset * max_results + ii;
//
//		__syncthreads();
//
//		if (r_iterator->dir != 1) continue;
//
//		end = r_iterator->end;
//		pos = r_iterator->pos;
//
//		if (pos > end) {
//			add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//			continue;
//		}
//
//		no_previous = true;
//		r_num_mismatches = r_iterator->num_mismatches-1;
//		if (r_num_mismatches>-1) {
//			last_err_pos  = r_iterator->err_pos[r_num_mismatches];
//			last_err_kind = r_iterator->err_kind[r_num_mismatches];
//			last_err_base = r_iterator->err_base[r_num_mismatches];
//		} else {
//			last_err_pos  = -10;
//			last_err_kind = 0;
//			last_err_base = (uint8_t) -1;
//		}
//
//		k = r_iterator->k;
//		l = r_iterator->l;
//
//		add_mismatch(r_iterator, DELETION, (uint8_t) -1, pos);
//
//		__syncthreads();
//
//		if (last_err_pos == pos - 1) { //Previous MISMATCH or DELETION
//
//			if (last_err_kind == MISMATCH) { //Previous MISMATCH
//
//				//Deletion
//				if (W[offset*MAXLINE+pos]!=last_err_base) {
//					change_result(r_iterator, k, l, pos+1);
//					add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//				}
//
//				for (uint8_t b=0;b<nA;b++) {
//
//					BWiterationGPU(k, l, k_aux, l_aux, b, Cshared, C1shared, O);
//
//					if (k_aux > l_aux) continue;
//
//					//Insertion
//					if (b!=W[offset*MAXLINE+last_err_pos]) {
//						change_result(r_iterator, k_aux, l_aux, pos);
//						modify_last_mismatch2(r_iterator, INSERTION, b);
//						add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//					}
//
//					//Mismatch
//					if (b!=W[offset*MAXLINE+pos]) {
//						change_result(r_iterator, k_aux, l_aux, pos+1);
//						modify_last_mismatch2(r_iterator, MISMATCH, b);
//						add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//					}
//
//				}
//
//				no_previous = false;
//
//			} else if (last_err_kind == DELETION) { //Previous DELETION
//
//				//Deletion
//				change_result(r_iterator, k, l, pos+1);
//				add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//				for (uint8_t b=0;b<nA;b++) {
//
//					BWiterationGPU(k, l, k_aux, l_aux, b, Cshared, C1shared, O);
//
//					if (k_aux > l_aux) continue;
//
//					// NO INSERTION
//
//					//Mismatch
//					if (b!=W[offset*MAXLINE+pos]) {
//
//						if (b!=W[offset*MAXLINE+last_err_pos]) {
//							change_result(r_iterator, k_aux, l_aux, pos+1);
//							modify_last_mismatch2(r_iterator, MISMATCH, b);
//							add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//						}
//
//					}
//
//				}
//
//				no_previous = false;
//
//			}
//
//		} else if (last_err_pos == pos) { //Previous INSERTION
//
//			//NO DELETION
//
//			for (uint8_t b=0;b<nA;b++) {
//
//				BWiterationGPU(k, l, k_aux, l_aux, b, Cshared, C1shared, O);
//
//				if (k_aux > l_aux) continue;
//
//				//Insertion
//				change_result(r_iterator, k_aux, l_aux, pos);
//				modify_last_mismatch2(r_iterator, INSERTION, b);
//				add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//				//Mismatch
//				if (b!=W[offset*MAXLINE+pos]) {
//
//					if (W[offset*MAXLINE+pos]!=last_err_base) {
//						r_iterator->pos = pos+1;
//						modify_last_mismatch1(r_iterator, MISMATCH);
//						add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//					}
//
//				}
//
//			}
//
//			no_previous = false;
//
//		}
//
//		if (no_previous) { //Previous MATCH
//
//			//Deletion
//			change_result(r_iterator, k, l, pos+1);
//			add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//			for (uint8_t b=0;b<nA;b++) {
//
//				BWiterationGPU(k, l, k_aux, l_aux, b, Cshared, C1shared, O);
//
//				if (k_aux > l_aux) continue;
//
//				//Insertion
//				change_result(r_iterator, k_aux, l_aux, pos);
//				modify_last_mismatch2(r_iterator, INSERTION, b);
//				add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//				if (b!=W[offset*MAXLINE+pos]) { //Mismatch
//					r_iterator->pos = pos+1;
//					modify_last_mismatch1(r_iterator, MISMATCH);
//					add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//				}
//
//			}
//
//		}
//
//	}
//
//	rl_next.num_results[offset] = rl_next_num_results;
//
//}
//
//void BWBranchPartialResultsForwardBlockedCPU(uintmax_t num_bloques, uintmax_t tam_bloques, uint8_t *W, vector *C, vector *C1, comp_matrix *O, blocked_results_lists rl_prev, blocked_results_lists rl_next, uintmax_t max_results, uint8_t nA) {
//
//	//#pragma omp parallel for
//	for(SA_TYPE offset=0; offset < num_bloques*tam_bloques; offset++) {
//
//		SA_TYPE k, l, k_aux, l_aux;
//		int16_t end, pos;
//		int16_t r_num_mismatches;
//		bool no_previous;
//		int16_t last_err_pos;
//		uint8_t last_err_kind;
//		uint8_t last_err_base;
//
//		result *r_iterator;
//
//		result *rl_next_results = rl_next.results + offset * max_results;
//		uintmax_t rl_next_num_results = 0;
//
//		for (uintmax_t ii=0; ii < rl_prev.num_results[offset]; ii++) {
//
//			r_iterator = rl_prev.results + offset * max_results + ii;
//
//			if (r_iterator->dir != 1) continue;
//
//			end = r_iterator->end;
//			pos = r_iterator->pos;
//
//			if (pos > end) {
//				add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//				continue;
//			}
//
//			no_previous = true;
//			r_num_mismatches = r_iterator->num_mismatches-1;
//			if (r_num_mismatches>-1) {
//				last_err_pos  = r_iterator->err_pos[r_num_mismatches];
//				last_err_kind = r_iterator->err_kind[r_num_mismatches];
//				last_err_base = r_iterator->err_base[r_num_mismatches];
//			} else {
//				last_err_pos  = -10;
//				last_err_kind = 0;
//				last_err_base = (uint8_t) -1;
//			}
//
//			k = r_iterator->k;
//			l = r_iterator->l;
//
//			add_mismatch(r_iterator, DELETION, (uint8_t) -1, pos);
//
//			if (last_err_pos == pos - 1) { //Previous MISMATCH or DELETION
//
//				if (last_err_kind == MISMATCH) { //Previous MISMATCH
//
//					//Deletion
//					if (W[offset*MAXLINE+pos]!=last_err_base) {
//						change_result(r_iterator, k, l, pos+1);
//						add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//					}
//
//					for (uint8_t b=0;b<nA;b++) {
//
//						BWiteration(k, l, k_aux, l_aux, b, C, C1, O);
//
//						if (k_aux > l_aux) continue;
//
//						//Insertion
//						if (b!=W[offset*MAXLINE+last_err_pos]) {
//							change_result(r_iterator, k_aux, l_aux, pos);
//							modify_last_mismatch2(r_iterator, INSERTION, b);
//							add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//						}
//
//						//Mismatch
//						if (b!=W[offset*MAXLINE+pos]) {
//							change_result(r_iterator, k_aux, l_aux, pos+1);
//							modify_last_mismatch2(r_iterator, MISMATCH, b);
//							add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//						}
//
//					}
//
//					no_previous = false;
//
//				} else if (last_err_kind == DELETION) { //Previous DELETION
//
//					//Deletion
//					change_result(r_iterator, k, l, pos+1);
//					add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//					for (uint8_t b=0;b<nA;b++) {
//
//						BWiteration(k, l, k_aux, l_aux, b, C, C1, O);
//
//						if (k_aux > l_aux) continue;
//
//						// NO INSERTION
//
//						//Mismatch
//						if (b!=W[offset*MAXLINE+pos]) {
//
//							if (b!=W[offset*MAXLINE+last_err_pos]) {
//								change_result(r_iterator, k_aux, l_aux, pos+1);
//								modify_last_mismatch2(r_iterator, MISMATCH, b);
//								add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//							}
//
//						}
//
//					}
//
//					no_previous = false;
//
//				}
//
//			} else if (last_err_pos == pos) { //Previous INSERTION
//
//				//NO DELETION
//
//				for (uint8_t b=0;b<nA;b++) {
//
//					BWiteration(k, l, k_aux, l_aux, b, C, C1, O);
//
//					if (k_aux > l_aux) continue;
//
//					//Insertion
//					change_result(r_iterator, k_aux, l_aux, pos);
//					modify_last_mismatch2(r_iterator, INSERTION, b);
//					add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//					//Mismatch
//					if (b!=W[offset*MAXLINE+pos]) {
//
//						if (W[offset*MAXLINE+pos]!=last_err_base) {
//							r_iterator->pos = pos+1;
//							modify_last_mismatch1(r_iterator, MISMATCH);
//							add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//						}
//
//					}
//
//				}
//
//				no_previous = false;
//
//			}
//
//			if (no_previous) { //Previous MATCH
//
//				//Deletion
//				change_result(r_iterator, k, l, pos+1);
//				add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//				for (uint8_t b=0;b<nA;b++) {
//
//					BWiteration(k, l, k_aux, l_aux, b, C, C1, O);
//
//					if (k_aux > l_aux) continue;
//
//					//Insertion
//					change_result(r_iterator, k_aux, l_aux, pos);
//					modify_last_mismatch2(r_iterator, INSERTION, b);
//					add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//
//					if (b!=W[offset*MAXLINE+pos]) { //Mismatch
//						r_iterator->pos = pos+1;
//						modify_last_mismatch1(r_iterator, MISMATCH);
//						add_resultGPU(r_iterator, rl_next_results, rl_next_num_results, max_results);
//					}
//
//				}
//
//			}
//
//		}
//
//		rl_next.num_results[offset] = rl_next_num_results;
//	}
//
//}
//
//void BWSearchGPU(uintmax_t num_bloques, uintmax_t tam_bloques, uint8_t *W, char *h_W, SA_TYPE nW, vector *C, vector *h_C, vector *C1, vector *h_C1, comp_matrix *O, comp_matrix *h_O, comp_matrix *Oi, comp_matrix *h_Oi, comp_vector *S, comp_vector *R, comp_vector *Si, comp_vector *Ri, blocked_results_lists *rl_prev_cpu, blocked_results_lists *rl_next_cpu, blocked_results_lists *rl_prev_i_cpu, blocked_results_lists *rl_next_i_cpu, blocked_results_lists *rl_final_cpu, blocked_results_lists *rl_prev_gpu, blocked_results_lists *rl_next_gpu, blocked_results_lists *rl_prev_i_gpu, blocked_results_lists *rl_next_i_gpu, blocked_results_lists *rl_final_gpu, int16_t fragsize, uintmax_t max_results) {
//
//	result r;
//
//	int16_t fragments = nW / fragsize;
//	int16_t half = fragments / 2;
//	if (fragments % 2) half++;
//	int err_count;
//
//	timevars();
//
//	//printf("\n----> Tamaño: %ju, Fragmentos: %ju Errores: %ju\n", nW, fragments, fragments-1);
//
//	/* //////////////////////////////FORWARD/////////////////////////////////////////// */
//
//	/* for (int16_t i = half-1; i>0; i--) { */
//
//	/* err_count = fragments-1; */
//	/* init_result(&r, 1); */
//	/* change_result(&r, 0, O->siz-2, fragsize*i); */
//	/* bound_result(&r, fragsize*i, fragsize*(i+1) - 1); */
//	/* BWExactSearchForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, nW, C->vector, C1->vector, *Oi, r, *rl_prev_gpu, max_results); */
//
//	/*   while (err_count > 0) { */
//
//	/*     BWExactPartialResultsForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1->vector, Oi, rl_prev, rl_next, rl_prev_i, fragsize, half-1); */
//	/*     BWChangeDirectionForwardBlockedGPU<<<num_bloques,tam_bloques>>>(Si, R, C, Oi, O, rl_prev_i, 0); */
//	/*     BWExactPartialResultsBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1->vector, O, rl_prev_i, rl_next_i, rl_final, fragsize, half-1); */
//	/*     BWBranchPartialResultsForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1->vector, Oi, rl_next, rl_prev); */
//	/*     BWBranchPartialResultsBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1->vector, O, rl_next_i, rl_prev_i); */
//
//	/*     err_count--; */
//
//	/*   } */
//
//	/*   BWExactFinalResultsForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1, Oi, rl_prev, rl_prev_i, fragsize, half-1); */
//	/*   BWChangeDirectionForwardBlockedGPU<<<num_bloques,tam_bloques>>>(Si, R, C, Oi, O, rl_prev_i, 0); */
//	/*   BWExactFinalResultsBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1, O, rl_prev_i, rl_final, fragsize, half-1); */
//
//	/* } */
//
//	///////BLOCK 0/////////////////////////////////////
//	err_count = fragments-1;
//
//	init_listsGPU<<<num_bloques,tam_bloques>>>(*rl_prev_gpu, *rl_next_gpu, *rl_prev_i_gpu, *rl_next_i_gpu, *rl_final_gpu);
//
//	init_result(&r, 1);
//	change_result(&r, 0, O->siz-2, 0);
//	bound_result(&r, 0, fragsize - 1);
//
//	// hipDeviceSynchronize();
//	// tic("Initial");
//	BWExactSearchForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, nW, C->vector, C1->vector, *Oi, r, *rl_prev_gpu, max_results);
//	// hipDeviceSynchronize();
//	// toc();
//
//	while (err_count > 0) {
//
//		// hipDeviceSynchronize();
//		// tic("Forward");
//		BWExactPartialResultsForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1->vector, *Oi, *rl_prev_gpu, *rl_next_gpu, *rl_final_gpu, fragsize, /*half-1*/0, max_results);
//		// hipDeviceSynchronize();
//		// toc();
//
//		// hipDeviceSynchronize();
//		// tic("To CPU");
//		// copy_blocked_results_list_cpu(rl_next_cpu, rl_next_gpu, max_results, num_bloques * tam_bloques);
//		// hipDeviceSynchronize();
//		// toc();
//		// hipDeviceSynchronize();
//		// tic("Branch");
//		// BWBranchPartialResultsForwardBlockedCPU(num_bloques, tam_bloques, h_W, h_C, h_C1, h_Oi, *rl_next_cpu, *rl_prev_cpu, max_results, nA);
//		// hipDeviceSynchronize();
//		// toc();
//		// hipDeviceSynchronize();
//		// tic("To GPU");
//		// copy_blocked_results_list_gpu(rl_prev_gpu, rl_prev_cpu, max_results, num_bloques * tam_bloques);
//		// hipDeviceSynchronize();
//		// toc();
//		if (err_count == 1) {
//
//			//  copy_blocked_results_list_cpu(rl_next_cpu, rl_next_gpu, max_results, num_bloques * tam_bloques);
//
//			//  hipDeviceSynchronize();
//			//  toc();
//			break;
//		}
//
//		// hipDeviceSynchronize();
//		// tic("Branch");
//		BWBranchPartialResultsForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1->vector, *Oi, *rl_next_gpu, *rl_prev_gpu, max_results, nA);
//		// hipDeviceSynchronize();
//		// toc();
//
//		err_count--;
//
//	}
//
//	hipDeviceSynchronize();
//	tic("FinalForward");
//	//BWBranchFinalResultsForwardBlockedCPU(num_bloques, tam_bloques, h_W, h_C, h_C1, h_Oi, *rl_next_cpu, *rl_final_cpu, fragsize, /*half-1*/0, max_results, nA);
//	BWBranchFinalResultsForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1->vector, *Oi, *rl_next_gpu, *rl_final_gpu, fragsize, /*half-1*/0, max_results, nA);
//	//BWExactFinalResultsForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1->vector, *Oi, *rl_prev_gpu, *rl_final_gpu, fragsize, /*half-1*/0, max_results);
//	hipDeviceSynchronize();
//	toc();
//
//	//copy_blocked_results_list_gpu(rl_final_gpu, rl_final_cpu, max_results, num_bloques * tam_bloques);
//
//
//	//////////////////////////////BACKWARD///////////////////////////////////////////
//
//	/* for (int16_t i = half; i<fragments-1; i++) { */
//
//	/* /\* printf("\n****BLOCK %d****\n", i); *\/ */
//	/* err_count = fragments-1; */
//
//	/* init_result(&r, 0); */
//	/* change_result(&r, 0, O->siz-2, fragsize*(i+1) - 1); */
//	/* bound_result(&r, fragsize*i, fragsize*(i+1) - 1); */
//	/* BWExactSearchBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, nW, C->vector, C1->vector, *O, r, *rl_prev_gpu, max_results); */
//
//	/*   while (err_count > 0) { */
//	/*     BWExactPartialResultsBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C, C1, O, rl_prev, rl_next, rl_prev_i, fragsize, 0); */
//	/*     BWChangeDirectionBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(S, Ri, C, O, Oi, rl_prev_i, nW-1); */
//	/*     BWExactPartialResultsForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C, C1, Oi, rl_prev_i, rl_next_i, rl_final, fragsize, 0); */
//	/*     BWBranchPartialResultsBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C, C1, O, rl_next, rl_prev); */
//	/*     BWBranchPartialResultsForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C, C1, Oi, rl_next_i, rl_prev_i); */
//	/*     err_count--; */
//	/*   } */
//
//	/*   BWExactFinalResultsBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C, C1, O, rl_prev, rl_prev_i, fragsize, 0); */
//	/*   BWChangeDirectionBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(S, Ri, C, O, Oi, rl_prev_i, nW-1); */
//	/*   BWExactFinalResultsForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C, C1, Oi, rl_prev_i, rl_final, fragsize, 0); */
//
//	/* } */
//
//	/* ///////BLOCK FRAGMENTS-1///////////////////////////////////// */
//	/* printf("\n****BLOCK %d****\n", fragments-1); */
//
//	/* err_count = fragments-1; */
//
//	/* init_result(&r, 0); */
//	/* change_result(&r, 0, O->siz-2, /\*fragsize*fragments - 1 Last block is larger*\/nW-1); */
//	/* bound_result(&r, fragsize*(fragments-1), /\*fragsize*fragments - 1 Last block is larger*\/nW-1); */
//	/* BWExactSearchBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, nW, C->vector, C1->vector, *O, r, *rl_prev_gpu, max_results); */
//
//	/* while (err_count > 0) { */
//	/*   BWExactPartialResultsBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1->vector, *O, *rl_prev_gpu, *rl_next_gpu, *rl_final_gpu, fragsize, 0, max_results); */
//	/*   BWBranchPartialResultsBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1->vector, *O, *rl_next_gpu, *rl_prev_gpu, max_results, nA); */
//	/*   err_count--; */
//	/* } */
//
//	/* BWExactFinalResultsBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, C->vector, C1->vector, *O, *rl_prev_gpu, *rl_final_gpu, fragsize, 0, max_results); */
//
//}
//
///* void BWExactSearchBackwardBlockedGPUWrapper(uintmax_t num_bloques, uintmax_t tam_bloques, uint8_t *W, SA_TYPE nW, vector *C, vector *C1, comp_matrix *O, result *r, blocked_results_lists *rl_prev_gpu, uintmax_t max_results) { */
///*   BWExactSearchBackwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, nW, C->vector, C1->vector, *O, *r, *rl_prev_gpu, max_results); */
///* } */
//
///* void BWExactSearchForwardBlockedGPUWrapper(uintmax_t num_bloques, uintmax_t tam_bloques, uint8_t *W, SA_TYPE nW, vector *C, vector *C1, comp_matrix *O, result *r, blocked_results_lists *rl_prev_gpu, uintmax_t max_results) { */
///*   BWExactSearchForwardBlockedGPU<<<num_bloques,tam_bloques>>>(W, nW, C->vector, C1->vector, *O, *r, *rl_prev_gpu, max_results); */
///* } */
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void BWExactSearchBackwardGPUWrapper(uintmax_t num_bloques, uintmax_t tam_bloques, uint8_t* W, uint64_t* nW, intmax_t* k, intmax_t* l, intmax_t k_ini, intmax_t l_ini, vector* C, vector* C1, comp_matrix* O) {
	BWExactSearchBackwardGPU<<<num_bloques,tam_bloques>>>(W, nW, k, l, k_ini, l_ini, C->vector, C1->vector, *O);
}

void BWExactSearchForwardGPUWrapper(uintmax_t num_bloques, uintmax_t tam_bloques, uint8_t* W, uint64_t* nW, intmax_t* k, intmax_t* l, intmax_t k_ini, intmax_t l_ini, vector* C, vector* C1, comp_matrix* O) {
	BWExactSearchForwardGPU<<<num_bloques,tam_bloques>>>(W, nW, k, l, k_ini, l_ini, C->vector, C1->vector, *O);
}

void BWExactSearchBackwardVectorGPUWrapper(uintmax_t num_bloques, uintmax_t tam_bloques, uint8_t* W, uint64_t* nW, intmax_t* k, intmax_t* l, intmax_t k_ini, intmax_t l_ini, vector* C, vector* C1, comp_matrix* O) {
	BWExactSearchBackwardVectorGPU<<<num_bloques,tam_bloques>>>(W, nW, k, l, k_ini, l_ini, C->vector, C1->vector, *O);
}

void BWExactSearchForwardVectorGPUWrapper(uintmax_t num_bloques, uintmax_t tam_bloques, uint8_t* W, uint64_t* nW, intmax_t* k, intmax_t* l, intmax_t k_ini, intmax_t l_ini, vector* C, vector* C1, comp_matrix* O) {
	BWExactSearchForwardVectorGPU<<<num_bloques,tam_bloques>>>(W, nW, k, l, k_ini, l_ini, C->vector, C1->vector, *O);
}
